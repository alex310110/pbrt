#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include "cuda_device.h"
#include <cmath>

void sort_on_device(std::vector<SampleData>& h_vec)
{
    // transfer data to the device
    thrust::device_vector<SampleData> d_vec(h_vec);

    // sort data on the device
    thrust::sort(d_vec.begin(), d_vec.end());

    // transfer data back to host
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
}

struct square_functor : public thrust::unary_function<SampleData,SampleData>
{
	__device__
	SampleData operator()(const SampleData &sd1) {
	SampleData s;
	for(int f=0; f<SampleData::getLastNormalizedOffset(); f++) {
		s[f] = sd1[f]*sd1[f];
	}
	return s;
}
};

struct norm : public thrust::unary_function<SampleData, SampleData>
{
	const SampleData mean, std;

	norm(SampleData mean, SampleData std) : mean(mean), std(std) {}

	__device__
	SampleData operator()(const SampleData &sd1) {
		SampleData s;
		s = sd1 - mean;
		s /= std;
	return s;
}
};

void normalize(std::vector<SampleData>& v) {
	// transfer data to the device
	thrust::device_vector<SampleData> d_v(v);

	SampleData mean, meanSquare, std;
	mean.reset(); meanSquare.reset();
	mean = thrust::reduce(d_v.begin(), d_v.end(), mean);
	meanSquare = thrust::transform_reduce(d_v.begin(), d_v.end(), square_functor(), meanSquare, thrust::plus<SampleData>());
	mean.divide(v.size());
	meanSquare.divide(v.size());
	for (int f=0; f < SampleData::getLastNormalizedOffset(); f++) {
		std[f] = sqrt(meanSquare[f] - mean[f]*mean[f]);
	}
	//TODO?
	thrust::transform(d_v.begin(), d_v.end(), d_v.begin(), norm(mean, std));

	thrust::copy(d_v.begin(), d_v.end(), v.begin());
}
